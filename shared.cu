/*
CUDA - generation and sum of arithmetic progression build of 10240000 elements a1=0 r=1 with shared memory
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__
void errorexit(const char *s) {
    printf("\n%s",s); 
    exit(EXIT_FAILURE);   
}

//elements generation
__global__ 
void calculate(long *result) {
    //allocate memory - size same as number of threads in block
   __shared__ long sresults[1024];  
    int counter;
    int my_index=blockIdx.x*blockDim.x+threadIdx.x;
    //write in shared memory element value for current thread
    sresults[threadIdx.x]=my_index;
    __syncthreads();
    //calculate sum of all elements of thread within same block using shared memory 
   for(counter=512;counter>0;counter/=2) {
      if (threadIdx.x<counter)
        sresults[threadIdx.x]=(sresults[threadIdx.x]+sresults[threadIdx.x+counter]);
      __syncthreads();      
    }

    //first thread in block write results of this block to global memory
    if (threadIdx.x==0) {
      result[blockIdx.x]=sresults[0];
    }
}


int main(int argc,char **argv) {

    long long result;
    int threadsinblock=1024;
    int blocksingrid=10000; 

    int size = threadsinblock*blocksingrid;
    
    //memory allocation on host
    long *hresults=(long*)malloc(blocksingrid*sizeof(long));
    if (!hresults) errorexit("Error allocating memory on the host");  
    
     //devie memory allocation (GPU)
    long *dresults=NULL;
    if (hipSuccess!=hipMalloc((void **)&dresults,blocksingrid*sizeof(long)))
      errorexit("Error allocating memory on the GPU");
    
    //call to GPU - kernel execution 
    calculate<<<blocksingrid,threadsinblock>>>(dresults);
    if (hipSuccess!=hipGetLastError())
      errorexit("Error during kernel launch");
     
      //getting results from GPU to host memory
    if (hipSuccess!=hipMemcpy(hresults,dresults,blocksingrid*sizeof(long),hipMemcpyDeviceToHost))
       errorexit("Error copying results");

    //calculate sum of all elements
    result=0;
    for(int i=0;i<blocksingrid;i++) {
      result = result + hresults[i];
    }

    printf("\nThe final result is %lld\n",result);

    //free memory
    free(hresults);
    if (hipSuccess!=hipFree(dresults))
      errorexit("Error when deallocating space on the GPU");

}
