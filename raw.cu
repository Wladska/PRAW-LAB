#include "hip/hip_runtime.h"
/*
CUDA - generation and sum of arithmetic progression build of 10240000 elements a1=0 r=1 without shared memory
*/
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

__host__
void errorexit(const char *s) {
    printf("\n%s",s);	
    exit(EXIT_FAILURE);	 	
}

//elements generation
__global__ 
void calculate(float *result) {
    int my_index = blockIdx.x*blockDim.x+threadIdx.x;
    result[my_index] = 1 / powf(2, my_index)
    __syncthreads ();

    for ( unsigned int s = 1; s < blockDim .x; s *= 2) {
      if ( my_index % (2 * s) == 0) {
        result[ my_index ] += result[ my_index + s ];
      }
      __syncthreads ();
    }
}


int main(int argc,char **argv) {

    float result;
    int threadsinblock=1024;
    int blocksingrid=10000;	

    int size = threadsinblock*blocksingrid;
    //memory allocation on host
    float *hresults=(float*)malloc(size*sizeof(float));
    if (!hresults) errorexit("Error allocating memory on the host");	

    float *dresults=NULL;
    //devie memory allocation (GPU)
    if (hipSuccess!=hipMalloc((void **)&dresults,size*sizeof(float)))
      errorexit("Error allocating memory on the GPU");

    //call to GPU - kernel execution 
    calculate<<<blocksingrid,threadsinblock>>>(dresults);
    if (hipSuccess!=hipGetLastError())
      errorexit("Error during kernel launch");
  
    //getting results from GPU to host memory
    if (hipSuccess!=hipMemcpy(hresults,dresults,size*sizeof(float),hipMemcpyDeviceToHost))
       errorexit("Error copying results");


    //calculate sum of all elements
    result=0;

    for(int i=0; i<blocksingrid; i++) {
      result = result + hresults[i * threadsinblock];
    }

    std::cout << "\nThe final result is " << result << std::endl;

    //free memory
    free(hresults);
    if (hipSuccess!=hipFree(dresults))
      errorexit("Error when deallocating space on the GPU");

}
